#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// Created by Yurii Shyrma on 02.01.2018
//

#include <ops/declarable/helpers/stack.h>
#include <helpers/ShapeUtils.h>
#include <array/ResultSet.h>
#include <cuda_exception.h>
#include <TAD.h>
#include <PointersManager.h>
#include <ConstantTadHelper.h>

namespace nd4j {
namespace ops {
namespace helpers {


	template <typename T>
	static __global__ void stackKernel(void** inputList, void** inputShapeList, int inputListLength, Nd4jLong arrLen, void* outputBuffer, Nd4jLong* tadShape, Nd4jLong *tadOffsets) {  //, Nd4jLong* tadShape, Nd4jLong* tadOffsets) {

		__shared__ int arrIdx, blocksPerArr;
		__shared__ T *z;
		__shared__ Nd4jLong *zShapeInfo, *xShapeInfo, arrLenPerBlock, start, end, offsetZ, zLength;

		if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuffer);
		}

		__syncthreads();

		for (int t = blockIdx.x; t < inputListLength; t += gridDim.x) {
            auto tZ = z + tadOffsets[t];
		    auto tX = reinterpret_cast<T*>(inputList[t]);
		    auto xShape = reinterpret_cast<Nd4jLong*>(inputShapeList[t]);

		    for (int e = threadIdx.x; e < arrLen; e += blockDim.x) {
		        tZ[shape::getIndexOffset(e, tadShape, arrLen)] = tX[shape::getIndexOffset(e, xShape, arrLen)];
            }
		}
	}
	///////////////////////////////////////////////////////////////////
	template <typename T>
	static void stack_(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim) {
		if(inArrs[0]->isScalar()) {

//#pragma omp parallel for
			for (size_t i = 0; i < inArrs.size(); ++i) {
                inArrs[i]->syncToHost();

                outArr->p(i, inArrs[i]->e<T>(0));
            }
			outArr->syncToDevice();
		}
		else {
			//Nd4jLong **dInShapeInfo;
			//void **dInBuffers;
			std::vector<void const*> inputList(inArrs.size());
			std::vector<Nd4jLong const*> inputShapeList(inArrs.size());
			auto stream = context->getCudaStream();

			for (size_t i = 0; i < inputList.size(); ++i) {
				inputList[i] = inArrs[i]->getSpecialBuffer();
				inputShapeList[i] = inArrs[i]->getSpecialShapeInfo();
			}

            std::vector<int> axis = ShapeUtils::evalDimsToExclude(outArr->rankOf(), {dim});


            auto packX = nd4j::ConstantTadHelper::getInstance()->tadForDimensions(outArr->getShapeInfo(), axis);


            PointersManager manager(context, "helpers::stack");
            auto dInBuffers = (void **) manager.replicatePointer(inputList.data(), inputList.size() * sizeof(Nd4jLong*));
            auto dInShapeInfo = (void **) manager.replicatePointer(inputShapeList.data(), inputShapeList.size() * sizeof(Nd4jLong*));

            dim3 launchDims(inArrs.size(), inArrs[0]->lengthOf(), 1024);

			stackKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>((void**)dInBuffers, (void**)dInShapeInfo, inputList.size(), inArrs[0]->lengthOf(), outArr->specialBuffer(), packX.specialShapeInfo(), packX.specialOffsets()); //, dTadShape, dTadOffsets);
            manager.synchronize();
		}
	}

	void stack(nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim) {
		BUILD_SINGLE_SELECTOR(outArr->dataType(), stack_, (context, inArrs, outArr, dim), LIBND4J_TYPES);
	}

	BUILD_SINGLE_TEMPLATE(template void stack_ , (nd4j::LaunchContext * context, const std::vector<NDArray*>& inArrs, NDArray* outArr, const int dim), LIBND4J_TYPES);

}
}
}

